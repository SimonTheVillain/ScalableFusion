#include "hip/hip_runtime.h"
#include <cuda/geom_update.h>

#include <cuda/vertex_update.h>
#include <cuda/std_tex_update.h>
#include <cuda/surface_read.h>
#include <cuda/xtion_camera_model.h>

//TODO: REMOVE DEBUG
#include <iostream>

using namespace std;
using namespace Eigen;

//We only should do a texture update for patches whose all neighbour patches are loaded.
//this means we do the vertex update for all patches that are loaded, but only do the textre update
//where all the neighbouring patches are present
//TODO: get rid of this integer debug value
int gpu::updateGeometry(const hipSurfaceObject_t geometry_input, //the sensor input adapted by standard deviations
                        int width, int height, //sensor resolution
                        const vector<gpu::UpdateDescriptor> &descriptors,
                        Vector4f cam_pos,
                        Matrix4f pose, // because we want the vertex position relative to the camera
                        Matrix4f proj_pose, //to get the position of the point on the image.
                        GpuVertex *vertices, Vector2f *tex_pos,
                        GpuTriangle *triangles, GpuPatchInfo *patch_infos) { //pointer to the geometric data

	if(descriptors.empty()) {
		return-1;
	}

	dim3 block(256);// using 1024); works on desktops but it is killing the tegra
	dim3 grid(descriptors.size());
	gpu::UpdateDescriptor *descs;
	hipMalloc(&descs, descriptors.size() * sizeof(gpu::UpdateDescriptor));
	hipMemcpy(descs, &descriptors[0], 
	           descriptors.size() * sizeof(gpu::UpdateDescriptor), 
	           hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	gpuErrchk(hipPeekAtLastError());

	//first we update the vertices
	vertexUpdate_kernel<<<grid, block>>>(geometry_input, //the sensor input adapted by standard deviations
	                                     width, height, //sensor resolution
	                                     descs,
	                                     cam_pos,
	                                     pose, // because we want the vertex position relative to the camera
	                                     proj_pose, //to get the position of the point on the image.
	                                     vertices, tex_pos,
	                                     triangles, patch_infos);

	hipDeviceSynchronize();//just for debug!!!
	gpuErrchk(hipPeekAtLastError());

	updateGeomTex_kernel<<<grid, block>>>(geometry_input, //the sensor input adapted by standard deviations
	                                      width, height, //sensor resolution
	                                      descs,
	                                      cam_pos,
	                                      pose, // because we want the vertex position relative to the camera
	                                      proj_pose, //to get the position of the point on the image.
	                                      vertices, tex_pos,
	                                      triangles, patch_infos);

	//then we update the texture.
	//why is this crashing????

	hipDeviceSynchronize();
	gpuErrchk(hipPeekAtLastError());
	hipFree(descs);

	return -1;
}

__global__
void checkVertexValidity_kernel(const hipSurfaceObject_t sensor_image,
                                int width, int height,
                                Matrix4f pose, Matrix4f proj_pose,
                                gpu::GeometryValidityChecks::VertexTask *tasks,
                                GpuVertex *vertices) {

	uint32_t k = blockIdx.x;
	gpu::GeometryValidityChecks::VertexTask task = tasks[k];

	uint32_t i = threadIdx.x;
	while(i < task.size) {
		GpuVertex vertex = vertices[i + task.start_source];

		//TODO: all the logic!!!!!
		//calculate position on the sensor
		Vector4f p = vertex.p;
		p[3] = 1;
		Vector4f p_cam = pose * p;
		float z = p_cam[2];

		Vector4f pos_on_sensor = proj_pose * p;

		float u = pos_on_sensor[0] / pos_on_sensor[3];
		float v = pos_on_sensor[1] / pos_on_sensor[3];

		float4 sensor = readSensor(u, v, sensor_image, width, height, 0.05); //threshold =0.1

		float threshold = xtionStdToThresholdSeg(sensor.y);//the second one is the surface

		if(z < (sensor.x - threshold) && !isnan(sensor.x)) {
			//invalidate vertex
			//for debug purposes do it in the source
			//TODO: do it at destRect
			vertices[i + task.start_source].valid = 0;
		}

		vertices[i + task.start_dest] = vertex;
		i += blockDim.x;
	}
}

void gpu::GeometryValidityChecks::checkVertexValidity(
		const hipSurfaceObject_t sensor, int width, int height, Matrix4f pose,
		Matrix4f proj_pose, vector<GeometryValidityChecks::VertexTask> tasks,
		GpuVertex *vertices) {

	if(tasks.empty()) {
		return;
	}

	dim3 block(256);
	dim3 grid(tasks.size());
	gpu::GeometryValidityChecks::VertexTask *gpu_tasks;

	hipMalloc(&gpu_tasks,
	           sizeof(gpu::GeometryValidityChecks::VertexTask) * tasks.size());

	hipMemcpy(gpu_tasks, &(tasks[0]),
	           sizeof(gpu::GeometryValidityChecks::VertexTask) * tasks.size(),
	           hipMemcpyHostToDevice);

	checkVertexValidity_kernel<<<grid, block>>>(sensor, width, height, pose,
	                                            proj_pose, gpu_tasks, vertices);

	hipFree(gpu_tasks);

	hipDeviceSynchronize();
	gpuErrchk(hipPeekAtLastError());
}

__global__
void checkTriangleValidity_kernel(
		gpu::GeometryValidityChecks::TriangleTask *tasks, 
		const hipSurfaceObject_t sensor, int width, int height, Matrix4f pose,
		Matrix4f proj_pose, GpuVertex *vertices, Vector2f *tex_pos, 
		GpuTriangle *triangles, GpuPatchInfo *patch_infos) {

}

void gpu::GeometryUpdate::calcCenterAndRadiusKernelCall_(
		dim3 grid, dim3 block, size_t bytes, CalcCenterTask *gpu_tasks,
		Vector4f *results) {
    hipDeviceSynchronize(); //TODO: REMOVE THESE DEBUG SYNCHRONIZATIONS!!!!
    gpuErrchk(hipPeekAtLastError());
    std::cout << "before running the calcCenter_kernel grid: " << grid.x << "block: " << block.x << std::endl;
	calcCenter_kernel<<<grid, block, bytes>>>(gpu_tasks, results);
    cout << "after running the calcCenter_kernel" << endl;
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    cout << "after running the calcCenter_kernel synchronization" << endl;
}

void gpu::GeometryValidityChecks::checkTriangleValidity(
		vector<TriangleTask> tasks, const hipSurfaceObject_t sensor,
		int width, int height, Matrix4f pose, Matrix4f proj_pose,
		GpuVertex *vertices, Vector2f *tex_pos, GpuTriangle *triangles,
		GpuPatchInfo *patch_infos) {

	assert(0); //this is not a priority yet.
	if(tasks.empty()) {
		return;
	}
	dim3 block(256);
	dim3 grid(tasks.size());
	GeometryValidityChecks::TriangleTask *gpu_tasks;

	hipMalloc(&gpu_tasks, 
	           sizeof(GeometryValidityChecks::TriangleTask) * tasks.size());

	hipMemcpy(gpu_tasks, &(tasks[0]),
	           sizeof(GeometryValidityChecks::TriangleTask) * tasks.size(),
	           hipMemcpyHostToDevice);

	checkTriangleValidity_kernel<<<grid, block>>>(gpu_tasks, sensor, width, 
	                                              height, pose, proj_pose,
	                                              vertices, tex_pos, triangles,
	                                              patch_infos);

	hipFree(gpu_tasks);

	hipDeviceSynchronize();
	gpuErrchk(hipPeekAtLastError());
}
